#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER ORasdadsa
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <hip/hip_runtime_api.h>
#include <chrono>
#include <vector>
#include <random>


//simple vector class with operator overloading for adding vectors and a randomized initialization:

class Vector {
public:
    Vector(int n) : data(n) {
        // Randomly initialize the vector with values between 0 and 100
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<> dist(0, 100);

        for (int i = 0; i < n; i++) {
            data[i] = dist(gen);
        }
    }

    Vector operator+(const Vector& other) const {
        Vector result(data.size());
        for (int i = 0; i < data.size(); i++) {
            result.data[i] = data[i] + other.data[i];
        }
        return result;
    }

    friend std::ostream& operator<<(std::ostream& os, const Vector& vec) {
        os << "[";
        for (int i = 0; i < vec.data.size(); i++) {
            os << vec.data[i];
            if (i != vec.data.size() - 1) {
                os << ", ";
            }
        }
        os << "]";
        return os;
    }

private:
    std::vector<int> data;
};

/////////////////////////////////////////////////////////////////
// Some utility code to define grid_stride_range
// Normally this would be in a header but it's here
// for didactic purposes. Uses
#include "range.hpp"
using namespace util::lang;

// type alias to simplify typing...
template <typename T>
using step_range = typename range_proxy<T>::step_range_proxy;

//compiler decides whether or not to actually inline the function based on a variety of factors such as code size and the optimization level

template <typename T>
inline __device__ step_range<T> grid_stride_range(T begin, T end) {
  begin += blockDim.x * blockIdx.x + threadIdx.x;
  return range(begin, end).step(gridDim.x * blockDim.x);
}

//simple operator overload - it allows you to add an integer to a grid_stride_range object 
//and return a new grid_stride_range object with its begin value increased by the integer value

//it's not actually used in any meaningful way in this code, it's just a representation of operator overloading
//lhs - left hand side, rhs - right hand side
template <typename T>
inline __device__ step_range<T> operator+(const step_range<T> &lhs, const int &rhs) {
  return range(lhs.begin() + rhs, lhs.end()).step(lhs.step());
}
/////////////////////////////////////////////////////////////////

// Overloading function count_if to take either a functor or a char value
// Simple SFINAE implementation - this function is enabled only when the type of Predicate is not char
template <typename T, typename Predicate,
			typename = typename std::enable_if<!std::is_same<Predicate, char>::value>::type>
__device__ void count_if(int *count, T *data, int n, Predicate p) {
  for (auto i : grid_stride_range(0, n)) {
    if (p(data[i])) atomicAdd(count, 1);
  }
}

template <typename T>
__device__ void count_if(int *count, T *data, int n, char value) {
  for (auto i : grid_stride_range(0, n)) {
    if (data[i] == value) atomicAdd(count, 1);
  }
}

// Use count_if with a lambda function that searches for x, y, z or w
// Note the use of range-based for loop and initializer_list inside the functor
// We use auto so we don't have to know the type of the functor or array
inline __global__ void xyzw_frequency(int *count, char *text, int n) {
  const char letters[]{'x', 'y', 'z', 'w'};

  count_if(count, text, n, [&](char c) {
    for (const auto x : letters)
      if (c == x) return true;
    return false;
  });
}

inline __global__ void xyzw_frequency_thrust_device(int *count, char *text, int n) {
  const char letters[]{'x', 'y', 'z', 'w'};
  *count = thrust::count_if(thrust::device, text, text + n, [=](char c) {
    for (const auto x : letters)
      if (c == x) return true;
    return false;
  });
}

// a bug in Thrust 1.8 causes warnings when this is uncommented
// so commented out by default -- fixed in Thrust master branch
#if 0 
void xyzw_frequency_thrust_host(int *count, char *text, int n)
{
  const char letters[] {'x', 'y', 'z', 'w'};
  *count = thrust::count_if(thrust::host, text, text+n, [&](char c) {
    for (const auto x : letters) 
      if (c == x) return true;
    return false;
  });
}
#endif

int main(int argc, char **argv) {
	
  std::chrono::steady_clock::time_point start=std::chrono::steady_clock::now();

  const char *filename = sdkFindFilePath("quovadis.txt", argv[0]);
  
  Vector v1(5), v2(5);

  int numBytes = 16 * 1048576;
  char *h_text = (char *)malloc(numBytes);

  // find first CUDA device
  int devID = findCudaDevice(argc, (const char **)argv);

  char *d_text;
  checkCudaErrors(hipMalloc((void **)&d_text, numBytes));

  FILE *fp = fopen(filename, "r");
  if (fp == NULL) {
    printf("Cannot find the input text file\n. Exiting..\n");
    return EXIT_FAILURE;
  }
  int len = (int)fread(h_text, sizeof(char), numBytes, fp);
  fclose(fp);
  std::cout << "Read " << len << " byte corpus from " << filename << std::endl;

  checkCudaErrors(hipMemcpy(d_text, h_text, len, hipMemcpyHostToDevice));

  int count = 0;
  int *d_count;
  checkCudaErrors(hipMalloc(&d_count, sizeof(int)));
  checkCudaErrors(hipMemset(d_count, 0, sizeof(int)));

  // Try uncommenting one kernel call at a time
  xyzw_frequency<<<8, 256>>>(d_count, d_text, len);
  xyzw_frequency_thrust_device<<<1, 1>>>(d_count, d_text, len);
  checkCudaErrors(
      hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));

  // xyzw_frequency_thrust_host(&count, h_text, len);
  
  std::chrono::steady_clock::time_point end=std::chrono::steady_clock::now();

  std::cout << "counted " << count
            << " instances of 'x', 'y', 'z', 'w' in \"" << filename << "\""
            << std::endl;
	

  std::cout << "v1: " << v1 << std::endl;
  std::cout << "v2: " << v2 << std::endl;
  Vector v3 = v1 + v2;
  std::cout << "v3 = v1 + v2: " << v3 << std::endl;
			
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  std::cout << "Execution time: " << duration.count() << " milliseconds" << std::endl;

  checkCudaErrors(hipFree(d_count));
  checkCudaErrors(hipFree(d_text));
  
  

  return EXIT_SUCCESS;
}
